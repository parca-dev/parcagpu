#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


// Simple kernel that does some work
__global__ void simpleKernel(int *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        // Simple computation to keep the kernel busy
        data[idx] = idx * idx + idx;
    }
}

// Another kernel for variety
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s - %s\n", msg, hipGetErrorString(err));
        exit(1);
    }
}

void runSimpleKernel() {
    const int n = 1024;
    const int size = n * sizeof(int);
    
    int *h_data = (int*)malloc(size);
    int *d_data;
    
    // Initialize host data
    for (int i = 0; i < n; i++) {
        h_data[i] = i;
    }
    
    // Allocate device memory
    checkCudaError(hipMalloc(&d_data, size), "hipMalloc failed");
    
    // Copy data to device
    checkCudaError(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice), "hipMemcpy H2D failed");
    
    // Launch kernel
    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
    
    printf("[App] Launching simpleKernel with grid=(%d,%d,%d), block=(%d,%d,%d)\n",
           gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z);
    
    simpleKernel<<<gridSize, blockSize>>>(d_data, n);
    checkCudaError(hipGetLastError(), "simpleKernel launch failed");
    
    // Wait for completion
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize failed");
    
    // Copy result back
    checkCudaError(hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost), "hipMemcpy D2H failed");
    
    // Cleanup
    hipFree(d_data);
    free(h_data);
}

void runVectorAdd() {
    const int n = 2048;
    const int size = n * sizeof(int);
    
    int *h_a = (int*)malloc(size);
    int *h_b = (int*)malloc(size);
    int *h_c = (int*)malloc(size);
    int *d_a, *d_b, *d_c;
    
    // Initialize host data
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    // Allocate device memory
    checkCudaError(hipMalloc(&d_a, size), "hipMalloc d_a failed");
    checkCudaError(hipMalloc(&d_b, size), "hipMalloc d_b failed");
    checkCudaError(hipMalloc(&d_c, size), "hipMalloc d_c failed");
    
    // Copy data to device
    checkCudaError(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice), "hipMemcpy H2D a failed");
    checkCudaError(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice), "hipMemcpy H2D b failed");
    
    // Launch kernel
    dim3 blockSize(512);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
    
    printf("[App] Launching vectorAdd with grid=(%d,%d,%d), block=(%d,%d,%d)\n",
           gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z);
    
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    checkCudaError(hipGetLastError(), "vectorAdd launch failed");
    
    // Wait for completion
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize failed");
    
    // Copy result back
    checkCudaError(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost), "hipMemcpy D2H c failed");
    
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
}

void runGraphLaunch() {
    const int n = 1024;
    const int size = n * sizeof(int);
    
    int *h_data = (int*)malloc(size);
    int *d_data;
    
    // Initialize host data
    for (int i = 0; i < n; i++) {
        h_data[i] = i;
    }
    
    // Allocate device memory
    checkCudaError(hipMalloc(&d_data, size), "hipMalloc failed");
    
    // Copy data to device
    checkCudaError(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice), "hipMemcpy H2D failed");
    
    // Create CUDA graph
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    
    checkCudaError(hipGraphCreate(&graph, 0), "hipGraphCreate failed");
    
    // Add kernel node to graph
    hipKernelNodeParams kernelParams = {0};
    kernelParams.func = (void*)simpleKernel;
    kernelParams.gridDim = dim3((n + 255) / 256, 1, 1);
    kernelParams.blockDim = dim3(256, 1, 1);
    kernelParams.sharedMemBytes = 0;
    
    void* kernelArgs[] = {(void*)&d_data, (void*)&n};
    kernelParams.kernelParams = kernelArgs;
    kernelParams.extra = NULL;
    
    hipGraphNode_t kernelNode;
    checkCudaError(hipGraphAddKernelNode(&kernelNode, graph, NULL, 0, &kernelParams), 
                   "hipGraphAddKernelNode failed");
    
    // Instantiate the graph
    checkCudaError(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0), 
                   "hipGraphInstantiate failed");
    
    printf("[App] Launching CUDA graph with kernel\n");
    
    // Launch the graph
    checkCudaError(hipGraphLaunch(graphExec, 0), "hipGraphLaunch failed");
    
    // Wait for completion
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize failed");
    
    // Copy result back
    checkCudaError(hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost), "hipMemcpy D2H failed");
    
    // Cleanup
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipFree(d_data);
    free(h_data);
}

int main() {
    printf("CUPTI Profiling Test Program\n");
    printf("============================\n");
    printf("This program runs real CUDA kernels to test CUPTI profiling\n\n");
    
    // Initialize CUDA driver API (required for CUPTI)
    hipError_t cuResult = hipInit(0);
    if (cuResult != hipSuccess) {
        fprintf(stderr, "hipInit failed: %d\n", cuResult);
        return 1;
    }
    
    // Initialize CUDA runtime
    checkCudaError(hipSetDevice(0), "hipSetDevice failed");
    
    int device;
    hipDeviceProp_t prop;
    checkCudaError(hipGetDevice(&device), "hipGetDevice failed");
    checkCudaError(hipGetDeviceProperties(&prop, device), "hipGetDeviceProperties failed");
    
    printf("Using device %d: %s\n", device, prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("Memory: %.2f GB\n\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
    
    // Run different compute patterns in a loop
    for (int iteration = 0; iteration < 10; iteration++) {
        printf("=== Iteration %d ===\n", iteration + 1);
        
        printf("Running simpleKernel...\n");
        runSimpleKernel();
        
        usleep(500000); // 500ms pause
        
        printf("Running vectorAdd...\n");
        runVectorAdd();
        
        usleep(500000); // 500ms pause
        
        printf("Running graph launch...\n");
        runGraphLaunch();
        
        usleep(500000); // 500ms pause
        
        printf("Iteration %d completed\n\n", iteration + 1);
    }
    
    printf("All iterations completed. Check profiling output for timing data.\n");
    
    return 0;
}